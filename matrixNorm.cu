/* Matrix normalization.
 * Compile with "gcc matrixNorm.c"
 */

 
#include <hip/hip_runtime.h>
#include <stdio.h>
 #include <stdlib.h>
 #include <time.h>
 #include <sys/time.h>
 #include <math.h>
 
 /* Program Parameters */
 #define MAXN 8000  /* Matrix size */
 
int BLOCK_SIZE = 256;

int N=6000;

 /* Matrices */
volatile float A[MAXN][MAXN], B[MAXN][MAXN];
float h_a[MAXN][MAXN], h_b[MAXN][MAXN];
 
 
 /* Initialize A and B*/
 void initialize_inputs() {
     int row, col;
     
     srand((unsigned)time(NULL));
     for (row = 0; row < N; row++) {
         for (col = 0; col < N; col++) {
             A[row][col] = (float)rand() / 32768.0;
             h_a[row][col] = A[row][col];
             B[row][col] = 0.0;
             h_b[row][col] = 0.0;

         }
     }
     
 }
 
 
 /* Kernel function */
 
 __global__ void matrixNorm(float *A, float *B, int n) {
    int col = blockIdx.x;
    int row, stride;
    int tid = threadIdx.x;
    float mu, sigma, partial=0; // Mean and Standard Deviation
    __shared__ float partials[1024], fullCol[MAXN];

    //set up partial sums and copy working column into shared memory
    for(row = threadIdx.x; row < n; row += blockDim.x){
        fullCol[row] = A[row*n + col];
        partial += fullCol[row];
    }
    partials[tid] = partial;
    __syncthreads();
    //reduction for sum
    for (stride = 1; stride < blockDim.x; stride *= 2) {
        if (tid % (2*stride) == 0){
            partials[tid] += partials[tid+stride];
        }
        __syncthreads();
    }
    //calculate mu, reset partial
    mu = partials[0]/n;
    partial = 0;


    //repeat for sigma
    for(row = threadIdx.x; row < n; row += blockDim.x){
        partial += powf(fullCol[row]-mu, 2.0);
    }
    partials[tid] = partial;
    __syncthreads();
    //reduction for variance * n
    for (stride = 1; stride < blockDim.x; stride *= 2) {
        if (tid % (2*stride) == 0){
            partials[tid] += partials[tid+stride];
        }
        __syncthreads();
    }
    //calculate mu
    sigma = partials[0]/n;
    sigma = sqrt(sigma);

    //use copied column to fill in B array
    for(row = threadIdx.x; row < n; row += blockDim.x){
        if (sigma == 0.0){
            B[row*n + blockIdx.x] = 0.0;
        }
        else{
            B[row*n + blockIdx.x] = (fullCol[row] -mu) / sigma;
        }
    }

}
 

void matrixNormSerial() {
    int row, col;
    float mu, sigma; // Mean and Standard Deviation
    
    printf("Computing Serially.\n");
    
    for (col=0; col < N; col++) {
        mu = 0.0;
        for (row=0; row < N; row++)
            mu += A[row][col];
        mu /= (float) N;
        sigma = 0.0;
        for (row=0; row < N; row++)
            sigma += powf(A[row][col] - mu, 2.0);
        sigma /= (float) N;
        sigma = sqrt(sigma);

        for (row=0; row < N; row++) {
            if (sigma == 0.0)
                B[row][col] = 0.0;
            else
                B[row][col] = (A[row][col] - mu) / sigma;
        }
    }
    
}
 
 
 int main(int argc, char **argv) {
     /* Timing variables */
     struct timeval start, stop;  /* Elapsed times using gettimeofday() */
     struct timezone tzdummy;
     unsigned long long runtime;
    
    
    
    if (argc == 3) {
        BLOCK_SIZE = atoi(argv[2]);
    }
    if(argc >=2){
        N = atoi(argv[1]);
        if (N < 1 || N > MAXN) {
            printf("N = %i is out of range.\n", N);
            exit(0);
        }
    }
    printf("Usage: %s [matrixDimension] [numThreadsPerBlock]\n",
    argv[0]);
        
    printf("Using defaults matrixDimension=%i, numThreadsPerBlock=%i\n", N, BLOCK_SIZE);
    



     /* Initialize A and B */
     initialize_inputs();
     
     
    // Allocate memory space on the device
    float *d_a, *d_b;
    hipMalloc((void **) &d_a, sizeof(float)*N*N);
    hipMalloc((void **) &d_b, sizeof(float)*N*N);

    // copy matrix A from host to device memory
    hipMemcpy(d_a, h_a, sizeof(float)*N*N, hipMemcpyHostToDevice);

    // some events to count the execution time
    hipEvent_t cstart, cstop;
    hipEventCreate(&cstart);
    hipEventCreate(&cstop);
    float gpu_elapsed_time_ms;

    dim3 dimGrid(N, 1, 1);
    dim3 dimBlock(BLOCK_SIZE, 1,1);


     /* Start Clock */
     printf("\n---------------------------------------------\n");
     printf("\nStarting clock.\n\n");
     gettimeofday(&start, &tzdummy);
    
    
    // Launch simple matrix multiplication kernel
    matrixNormSerial();

    /* Stop Clock */
    gettimeofday(&stop, &tzdummy);
    runtime = (unsigned long long)(stop.tv_sec - start.tv_sec) * 1000000 + (stop.tv_usec - start.tv_usec); 

     
     /* Display timing results */
     printf("Runtime = %g ms.\n", (float)runtime/(float)1000);
     printf("\nStopped clock.");
     printf("\n---------------------------------------------\n");
    

     /* Start Clock */
     printf("\n---------------------------------------------\n");
     printf("\nStarting Cuda clock.\n\n");
     hipEventRecord(cstart, 0);

     matrixNorm<<<dimGrid, dimBlock>>>(d_a, d_b, N);  
    // start to count execution time of GPU Kernel 
     hipEventRecord(cstop, 0);
     hipEventSynchronize(cstop);
 
     // Transfer results from device to host
     hipMemcpy(h_b, d_b, sizeof(float)*N*N, hipMemcpyDeviceToHost);
    // compute time elapse on GPU computing
    hipEventElapsedTime(&gpu_elapsed_time_ms, cstart, cstop);
    printf("Time elapsed on matrix norm on GPU: %f ms.\n\n", gpu_elapsed_time_ms);
    printf("Runtime = %g ms.\n", (float)gpu_elapsed_time_ms);
    printf("\nStopped clock.");
    printf("\n---------------------------------------------\n");

    hipFree(d_a);
    hipFree(d_b);
    hipHostFree(h_a);
    hipHostFree(h_b);

    int i;
    printf("Spot check for correctness on row 100, cols 0-9: \n");
    for(i=0; i < 10; i++){
        printf("B: %5.2f  b_h: %5.2f\n", B[100][i], h_b[100][i]);
    }


     exit(0);
 }